// UCSD Phys244
// Spring 2021
// Andreas Goetz (agoetz@sdsc.edu)

// CUDA program to square matrix elements in parallel on the GPU
//


#include <hip/hip_runtime.h>
#include<stdio.h>

// define matrix size, number of blocks NBL and threads per block TPB
#define NROW 32768
#define NCOL 16384
#define NBLX 32
#define NBLY 32
#define TPBX 16
#define TPBY 16

//
// CUDA device function that squares elements of a 2D array
//
__global__ void square(long int *arr, size_t maxrow, size_t maxcol){

  size_t rowinit = threadIdx.x + blockDim.x * blockIdx.x;
  size_t colinit = threadIdx.y + blockDim.y * blockIdx.y;
  size_t rowstride = gridDim.x * blockDim.x;
  size_t colstride = gridDim.y * blockDim.y;
  size_t pos;

  // operate on all submatrices
  for (size_t row = rowinit; row < maxrow; row += rowstride) {
    for (size_t col = colinit; col < maxcol; col += colstride) {
      pos = row*maxcol + col;
      arr[pos] *= arr[pos];
    }
  }

}

//
// main program
//
int main(void){

  long int h_a[NROW][NCOL];
  long int *d_a;
  size_t size = NROW * NCOL * sizeof(long int);
  int err;

  // allocate device memory
  hipMalloc((void **)&d_a, size);

  // initialize matrix
  for (size_t i=0; i<NROW; i++){
    for (size_t j=0; j<NCOL; j++){
      h_a[i][j] = i+j;
      // printf("Element (%d,%d) = %d\n",i,j,h_a[i][j]);
    }
  }

  // copy input data to device
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);

  // add vectors by launching a sufficient number of blocks of the add() kernel
  printf("\nLaunching kernel to square matrix elements...\n");
  printf("Matrix elements   = %d x %d = %d\n",NROW,NCOL,NROW*NCOL);
  printf("Blocks            = %d x %d = %d\n",NBLX,NBLY,NBLX*NBLY);
  printf("Threads per block = %d x %d = %d\n",TPBX,TPBY,TPBX*TPBY);
  printf("Kernel copies     = %d\n",NBLX*NBLY*TPBX*TPBY);
  square<<<dim3(NBLX,NBLY),dim3(TPBX,TPBY)>>>(d_a, NROW, NCOL);

  // copy results back to host
  hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);

  // deallocate memory
  hipFree(d_a);

  // check results
  err = 0;
  for (size_t i=0; i<NROW; i++){
    for (size_t j=0; j<NCOL; j++){
      if (h_a[i][j] != (i+j)*(i+j)) err += 1;
      //printf("Element (%d,%d) = %d\n",i,j,h_a[i][j]);
    }
  }
  if (err != 0){
    printf("\n Error, %d elements do not match!\n\n", err);
  } else {
    printf("\n Success! All elements match.\n\n");
  }

  return 0;

}
